#include "hip/hip_runtime.h"
﻿//--------------------------------------------------------------------------------------------------------------------------------------------------
// Project : Real-time-Audio-Feature-Extraction-with-CUDA-for-Speech-Recognition
// Implement a system that uses CUDA to accelerate the extraction of audio features (e.g., MFCCs) from live audio streams.
// Author: Arsheya Raj
// Date: 8th April 2025
//--------------------------------------------------------------------------------------------------------------------------------------------------
//
//  Develop a system that utilizes CUDA to accelerate the extraction of audio features like Mel-frequency cepstral coefficients (MFCCs)
//  from live audio streams. This enables real-time speech recognition by providing pre-processed audio data to a speech-to-text model
//  running on local hardware.
//
//--------------------------------------------------------------------------------------------------------------------------------------------------

#include "hip/hip_runtime.h"
#include ""
#include <math.h>

#include <stdio.h>

extern "C" {
    __global__ void apply_hamming(float* signal, int len) {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if (i < len) {
            signal[i] *= 0.54f - 0.46f * cosf(2.0f * 3.14159265f * i / (len - 1));
        }
    }

    __global__ void dummy_fft(float* signal, int len) {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if (i < len) {
            signal[i] = logf(1 + fabsf(signal[i]));
        }
    }

    __declspec(dllexport)
        void compute_mfcc(float* signal, int len, int sample_rate, float* mfcc_out) {
        float* d_signal;
        hipMalloc((void**)&d_signal, len * sizeof(float));
        hipMemcpy(d_signal, signal, len * sizeof(float), hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (len + blockSize - 1) / blockSize;
        apply_hamming << <numBlocks, blockSize >> > (d_signal, len);
        dummy_fft << <numBlocks, blockSize >> > (d_signal, len);

        float temp[13];
        hipMemcpy(temp, d_signal, 13 * sizeof(float), hipMemcpyDeviceToHost);

        for (int i = 0; i < 13; i++)
            mfcc_out[i] = temp[i];

        hipFree(d_signal);
    }
}